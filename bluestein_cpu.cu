#include "hip/hip_runtime.h"
#include "common.hu"
#include <cmath>

#define PI 3.141592654f

float calc_w(float r, float n)
{
  return (- (2 * PI * r) / n);
}

float bluestein(hipEvent_t start, hipEvent_t stop,
                DATA_TYPE *dev_data, DATA_TYPE *dev_middle)
{
    float time;

    size_t N = DATA_SIZE;
    size_t M = pow(2.0, ceil(log2((double)(N - 1)) + 1));

    hipfftHandle plan;
    cufftCheckReturn(hipfftCreate(&plan));
    long long len = M;
    size_t ws = 0;

    cufftCheckReturn(
        hipfftXtMakePlanMany(
            plan, 1,  &len,
            NULL, 1, 1, HIP_C_32F,
            NULL, 1, 1, HIP_C_32F,
            1, &ws, HIP_C_32F));

    cudaCheckReturn(hipDeviceSynchronize());
    cudaCheckReturn(hipEventRecord(start));

    DATA_TYPE *h, *hh, *x, *y, *CY, *hCZ, *CZ, *CX;

    cudaCheckReturn(hipHostMalloc(&h,     M * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipHostMalloc(&hh,    M * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipHostMalloc(&x,     N * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipHostMalloc(&y,     M * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipHostMalloc(&CY,    M * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipHostMalloc(&hCZ,   M * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipHostMalloc(&CZ,    M * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipHostMalloc(&CX,    N * sizeof(DATA_TYPE)));

    for (int l = 0; l <= N-1; l++) {
      float p = calc_w((- 1 / (float)2) * (l * l), N);;
      h[l].x = cosf(p);
      h[l].y = sinf(p);
    }
    for (int l = M - N + 1; l <= M - 1; l++) {
      h[l] = h[M - l];
    }
    for (int l = N; l <= M - N; l++) {
      h[l].x = 0.f;
      h[l].y = 0.f;
    }

    for (int r = 0; r <= M - 1; r++) {
      hh[r].x = 0.f;
      hh[r].y = 0.f;

      for (int l = 0; l <= M - 1; l++) {
        float p = calc_w(r * l, M);
        hh[r].x += h[l].x * cosf(p) - h[l].y * sinf(p);
        hh[r].y += h[l].x * sinf(p) + h[l].y * cosf(p);
      }
    }

    cudaCheckReturn(hipMemcpy(x, dev_data, N * sizeof(DATA_TYPE),
                               hipMemcpyDeviceToHost));

    for (int l = 0; l <= N - 1; l++) {
      float p = calc_w((1 / (float)2) * l * l, N);
      y[l].x = x[l].x * cosf(p) - x[l].y * sinf(p);
      y[l].y = x[l].x * sinf(p) + x[l].y * cosf(p);
    }
    for (int l = N; l <= M - 1; l++) {
      y[l].x = 0.f;
      y[l].y = 0.f;
    }

    for (int r = 0; r <= M - 1; r++) {
      CY[r].x = 0.f;
      CY[r].y = 0.f;

      for (int l = 0; l <= M - 1; l++) {
        float p = calc_w(r * l, M);
        CY[r].x += y[l].x * cosf(p) - y[l].y * sinf(p);
        CY[r].y += y[l].x * sinf(p) + y[l].y * cosf(p);
      }
    }

    for (int r = 0; r <= M - 1; r++) {
      hCZ[r].x = CY[r].x * hh[r].x - CY[r].y * hh[r].y;
      hCZ[r].y = CY[r].x * hh[r].y + CY[r].y * hh[r].x;
    }


    for (int r = 0; r <= M - 1; r++) {
      CZ[r].x = 0.f;
      CZ[r].y = 0.f;

      for (int l = 0; l <= M - 1; l++) {
        float p = calc_w(- r * l, M);
        CZ[r].x += hCZ[l].x * cosf(p) - hCZ[l].y * sinf(p);
        CZ[r].y += hCZ[l].x * sinf(p) + hCZ[l].y * cosf(p);
      }

      CZ[r].x /= M;
      CZ[r].y /= M;
    }

    for (int r = 0; r <= N - 1; r++) {
      float p = calc_w((1 / (float)2) * r * r, N);
      CX[r].x = CZ[r].x * cosf(p) - CZ[r].y * sinf(p);
      CX[r].y = CZ[r].x * sinf(p) + CZ[r].y * cosf(p);
    }

    cudaCheckReturn(hipMemcpy(dev_middle, CX, N * sizeof(DATA_TYPE),
                               hipMemcpyHostToDevice));

    // cufftCheckReturn(hipfftXtExec(plan, dev_data, dev_middle, HIPFFT_FORWARD));

    cudaCheckReturn(hipEventRecord(stop));
    cudaCheckReturn(hipEventSynchronize(stop));

    cudaCheckReturn(hipEventElapsedTime(&time, start, stop));

    return time;
}

std::vector<float> benchmark(DATA_TYPE *output,
                             DATA_TYPE *data,
                             hipEvent_t start, hipEvent_t stop)
{
    DATA_TYPE *dev_output, *dev_middle, *dev_data, *middle;
    std::vector<float> time(2);
    
    size_t N = DATA_SIZE;
    size_t M = pow(2.0, ceil(log2((double)(N - 1)) + 1));

    /*
      Setup
    */
    cudaCheckReturn(hipHostMalloc(&middle, M * sizeof(DATA_TYPE)));

    cudaCheckReturn(hipMalloc(&dev_data,   M * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipMalloc(&dev_middle, M * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipMalloc(&dev_output, N * sizeof(DATA_TYPE)));

    cudaCheckReturn(hipMemcpy(dev_data, data, N * sizeof(DATA_TYPE),
                               hipMemcpyHostToDevice));

    hipfftHandle plan;
    cufftCheckReturn(hipfftCreate(&plan));
    long long len = N;
    size_t ws = 0;

    cufftCheckReturn(
        hipfftXtMakePlanMany(
            plan, 1,  &len,
            NULL, 1, 1, HIP_C_32F,
            NULL, 1, 1, HIP_C_32F,
            1, &ws, HIP_C_32F));

    /*
      FFT
    */
    time[0] = bluestein(start, stop, dev_data, dev_middle);
    cudaCheckKernel();

    /*
      Scaling
    */
    cudaCheckReturn(hipMemcpy(middle, dev_middle, N * sizeof(DATA_TYPE),
                               hipMemcpyDeviceToHost));

    for (size_t i = 0; i < DATA_SIZE; i++) {
        float2 m = middle[i];
        m.x /= DATA_SIZE;
        m.y /= DATA_SIZE;
        middle[i] = m;
    }    

    cudaCheckReturn(hipMemcpy(dev_middle, middle, N * sizeof(DATA_TYPE),
                               hipMemcpyHostToDevice));

    /*
      IFFT
    */
    cudaCheckReturn(hipDeviceSynchronize());
    cudaCheckReturn(hipEventRecord(start));

    cufftCheckReturn(hipfftXtExec(plan, dev_middle, dev_output, HIPFFT_BACKWARD));

    cudaCheckReturn(hipEventRecord(stop));
    cudaCheckReturn(hipEventSynchronize(stop));
    cudaCheckKernel();

    cudaCheckReturn(hipEventElapsedTime(&time[1], start, stop));

    /*
      Close
    */
    cufftCheckReturn(hipfftDestroy(plan));

    cudaCheckReturn(hipMemcpy(output, dev_output, N * sizeof(DATA_TYPE),
                               hipMemcpyDeviceToHost));

    cudaCheckReturn(hipHostFree(middle));

    cudaCheckReturn(hipFree(dev_output));
    cudaCheckReturn(hipFree(dev_middle));
    cudaCheckReturn(hipFree(dev_data));

    return time;
}
