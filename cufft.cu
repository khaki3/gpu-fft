#include "common.hu"

typedef half2 ftype;
long long sig_size = 1<<23;

int main ()
{
  ftype *h_idata = (ftype *)malloc(sig_size*sizeof(ftype));
  ftype *d_idata;
  ftype *d_odata;
  hipMalloc(&d_idata, sizeof(ftype)*sig_size);
  hipMalloc(&d_odata, sizeof(ftype)*sig_size);

  hipfftHandle plan;
  cufftCheckReturn(hipfftCreate(&plan));
  size_t ws = 0;

  cufftCheckReturn(hipfftXtMakePlanMany(plan, 1,  &sig_size, NULL, 1, 1, HIP_C_16F, NULL, 1, 1, HIP_C_16F, 1, &ws, HIP_C_16F));
  cufftCheckReturn(hipfftXtExec(plan, d_idata, d_odata, HIPFFT_FORWARD)); // warm-up

  hipEvent_t start, stop;
  hipEventCreate(&start); hipEventCreate(&stop);
  hipEventRecord(start);

  cufftCheckReturn(hipfftXtExec(plan, d_idata, d_odata, HIPFFT_FORWARD));

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float et;

  hipEventElapsedTime(&et, start, stop);
  printf("forward FFT time for %ld samples: %fms\n", sig_size, et);

  return 0;
}
