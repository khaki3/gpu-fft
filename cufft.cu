#include "hip/hip_runtime.h"
#include "common.hu"

std::vector<float> benchmark(DATA_TYPE *output,
                             DATA_TYPE *data,
                             hipEvent_t start, hipEvent_t stop)
{
    DATA_TYPE *dev_output, *dev_middle, *dev_data, *middle;
    std::vector<float> time(2);

    /*
      Setup
    */
    cudaCheckReturn(hipHostMalloc(&middle, DATA_SIZE * sizeof(DATA_TYPE)));

    cudaCheckReturn(hipMalloc(&dev_data,   DATA_SIZE * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipMalloc(&dev_middle, DATA_SIZE * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipMalloc(&dev_output, DATA_SIZE * sizeof(DATA_TYPE)));

    cudaCheckReturn(hipMemcpy(dev_data, data, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyHostToDevice));

    hipfftHandle plan;
    cufftCheckReturn(hipfftCreate(&plan));
    long long len = DATA_SIZE;
    size_t ws = 0;

    cufftCheckReturn(
        hipfftXtMakePlanMany(
            plan, 1,  &len,
            NULL, 1, 1, HIP_C_32F,
            NULL, 1, 1, HIP_C_32F,
            1, &ws, HIP_C_32F));

    /*
      FFT
    */
    cudaCheckReturn(hipDeviceSynchronize());
    cudaCheckReturn(hipEventRecord(start));

    cufftCheckReturn(hipfftXtExec(plan, dev_data, dev_middle, HIPFFT_FORWARD));

    cudaCheckReturn(hipEventRecord(stop));
    cudaCheckReturn(hipEventSynchronize(stop));
    cudaCheckKernel();

    cudaCheckReturn(hipEventElapsedTime(&time[0], start, stop));

    /*
      Scaling
    */
    cudaCheckReturn(hipMemcpy(middle, dev_middle, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyDeviceToHost));

    for (size_t i = 0; i < DATA_SIZE; i++) {
        float2 m = middle[i];
        m.x /= DATA_SIZE;
        m.y /= DATA_SIZE;
        middle[i] = m;
    }    

    cudaCheckReturn(hipMemcpy(dev_middle, middle, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyHostToDevice));

    /*
      IFFT
    */
    cudaCheckReturn(hipDeviceSynchronize());
    cudaCheckReturn(hipEventRecord(start));

    cufftCheckReturn(hipfftXtExec(plan, dev_middle, dev_output, HIPFFT_BACKWARD));

    cudaCheckReturn(hipEventRecord(stop));
    cudaCheckReturn(hipEventSynchronize(stop));
    cudaCheckKernel();

    cudaCheckReturn(hipEventElapsedTime(&time[1], start, stop));

    /*
      Close
    */
    cufftCheckReturn(hipfftDestroy(plan));

    cudaCheckReturn(hipMemcpy(output, dev_output, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyDeviceToHost));

    cudaCheckReturn(hipHostFree(middle));

    cudaCheckReturn(hipFree(dev_output));
    cudaCheckReturn(hipFree(dev_middle));
    cudaCheckReturn(hipFree(dev_data));

    return time;
}
