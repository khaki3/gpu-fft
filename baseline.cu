#include "hip/hip_runtime.h"
#include "common.hu"

#define HIP_PI_F 3.141592654f

__global__ void fft(DATA_TYPE *output, DATA_TYPE *data)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;

    float real = 0.0;
    float imag = 0.0;

    if (id < DATA_SIZE) {
        float pow = 2 * HIP_PI_F * id / (float)DATA_SIZE;

        for (size_t i = 0; i < DATA_SIZE; i++) {
            /*
                r * cos(2pkl/n) + i * sin(2pkl/n)
              - r * sin(2pkl/n) + i * con(2pkl/n)
            */
            DATA_TYPE d = data[i];
            float powh = fmodf(i * pow, 2 * HIP_PI_F);

            real +=   d.x * cosf(powh) + d.y * sinf(powh);
            imag += - d.x * sinf(powh) + d.y * cosf(powh);
        }

        output[id] = make_float2(real, imag);
    }
}

__global__ void ifft(DATA_TYPE *output, DATA_TYPE *data)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;

    float real = 0.0;
    float imag = 0.0;

    if (id < DATA_SIZE) {
        float pow = 2 * HIP_PI_F * id / (float)DATA_SIZE;

        for (size_t i = 0; i < DATA_SIZE; i++) {
            /*
              r * cos(2pkl/n) - i * sin(2pkl/n)
              r * sin(2pkl/n) + i * con(2pkl/n)
            */
            DATA_TYPE d = data[i];
            float powh = fmodf(i * pow, 2 * HIP_PI_F);

            real += d.x * cosf(powh) - d.y * sinf(powh);
            imag += d.x * sinf(powh) + d.y * cosf(powh);
        }

        output[id] = make_float2(real, imag);
    }
}

std::vector<float> benchmark(DATA_TYPE *output,
                             DATA_TYPE *data,
                             hipEvent_t start, hipEvent_t stop)
{
    DATA_TYPE *dev_output, *dev_middle, *dev_data, *middle;
    std::vector<float> time(2);

    /*
      Setup
    */
    cudaCheckReturn(hipHostMalloc(&middle, DATA_SIZE * sizeof(DATA_TYPE)));

    cudaCheckReturn(hipMalloc(&dev_data,   DATA_SIZE * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipMalloc(&dev_middle, DATA_SIZE * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipMalloc(&dev_output, DATA_SIZE * sizeof(DATA_TYPE)));

    cudaCheckReturn(hipMemcpy(dev_data, data, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyHostToDevice));

    /*
      FFT
    */
    cudaCheckReturn(hipDeviceSynchronize());
    cudaCheckReturn(hipEventRecord(start));

    fft<<<DATA_SIZE / 256, 256>>>(dev_middle, dev_data);

    cudaCheckReturn(hipEventRecord(stop));
    cudaCheckReturn(hipEventSynchronize(stop));
    cudaCheckKernel();

    cudaCheckReturn(hipEventElapsedTime(&time[0], start, stop));

    /*
      Scaling
    */
    cudaCheckReturn(hipMemcpy(middle, dev_middle, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyDeviceToHost));

    for (size_t i = 0; i < DATA_SIZE; i++) {
        float2 m = middle[i];
        m.x /= DATA_SIZE;
        m.y /= DATA_SIZE;
        middle[i] = m;
    }    

    cudaCheckReturn(hipMemcpy(dev_middle, middle, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyHostToDevice));

    /*
      IFFT
    */
    cudaCheckReturn(hipDeviceSynchronize());
    cudaCheckReturn(hipEventRecord(start));

    ifft<<<DATA_SIZE / 256, 256>>>(dev_output, dev_middle);

    cudaCheckReturn(hipEventRecord(stop));
    cudaCheckReturn(hipEventSynchronize(stop));
    cudaCheckKernel();

    cudaCheckReturn(hipEventElapsedTime(&time[1], start, stop));

    /*
      Close
    */
    cudaCheckReturn(hipMemcpy(output, dev_output, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyDeviceToHost));

    cudaCheckReturn(hipHostFree(middle));

    cudaCheckReturn(hipFree(dev_output));
    cudaCheckReturn(hipFree(dev_middle));
    cudaCheckReturn(hipFree(dev_data));

    return time;
}
