#include "hip/hip_runtime.h"
#include "common.hu"

#define HIP_PI_F 3.141592654f

__global__ void fft(DATA_TYPE *output, DATA_TYPE *data)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;

    float2 tmp = make_float2(0.0, 0.0);

    if (id < DATA_SIZE) {
        float pow = 2 * HIP_PI_F * id / (float)DATA_SIZE;
        DATA_TYPE m = __floats2half2_rn(1.0, -1.0);

        for (size_t i = 0; i < DATA_SIZE; i++) {
            /*
                r * cos(2pkl/n) + i * sin(2pkl/n)
              - r * sin(2pkl/n) + i * con(2pkl/n)
            */

            DATA_TYPE d = data[i];
            DATA_TYPE powh = __float2half2_rn(fmodf(i * pow, 2 * HIP_PI_F));

            /* 
               c   = (r * cos,   i * cos)
               smr = (i * sin, - r * sin)
            */
            DATA_TYPE c   = __hmul2(d, h2cos(powh));
            DATA_TYPE smr = __hmul2(__lowhigh2highlow(__hmul2(d, h2sin(powh))), m);

            float2 f = __half22float2(__hadd2(c, smr));
            tmp.x += f.x;
            tmp.y += f.y;
        }

        output[id] = __float22half2_rn(tmp);
    }
}

__global__ void ifft(DATA_TYPE *output, DATA_TYPE *data)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;

    float2 tmp = make_float2(0.0, 0.0);

    if (id < DATA_SIZE) {
        float pow = 2 * HIP_PI_F * id / (float)DATA_SIZE;
        DATA_TYPE m  = __floats2half2_rn(-1.0, 1.0);

        for (size_t i = 0; i < DATA_SIZE; i++) {
            /*
              r * cos(2pkl/n) - i * sin(2pkl/n)
              r * sin(2pkl/n) + i * con(2pkl/n)
            */

            DATA_TYPE d = data[i];
            DATA_TYPE powh = __float2half2_rn(fmodf(i * pow, 2 * HIP_PI_F));

            /* 
               c   = (  r * cos, i * cos)
               smr = (- i * sin, r * sin)
            */
            DATA_TYPE c   = __hmul2(d, h2cos(powh));
            DATA_TYPE smr = __hmul2(__lowhigh2highlow(__hmul2(d, h2sin(powh))), m);

            float2 f = __half22float2(__hadd2(c, smr));
            tmp.x += f.x;
            tmp.y += f.y;
        }

        output[id] = __float22half2_rn(tmp);
    }
}

std::vector<float> benchmark(DATA_TYPE *output,
                             DATA_TYPE *data,
                             hipEvent_t start, hipEvent_t stop)
{
    DATA_TYPE *dev_output, *dev_middle, *dev_data, *middle;
    std::vector<float> time(2);

    /*
      Setup
    */
    cudaCheckReturn(hipHostMalloc(&middle, DATA_SIZE * sizeof(DATA_TYPE)));

    cudaCheckReturn(hipMalloc(&dev_data,   DATA_SIZE * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipMalloc(&dev_middle, DATA_SIZE * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipMalloc(&dev_output, DATA_SIZE * sizeof(DATA_TYPE)));

    cudaCheckReturn(hipMemcpy(dev_data, data, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyHostToDevice));

    /*
      FFT
    */
    cudaCheckReturn(hipDeviceSynchronize());
    cudaCheckReturn(hipEventRecord(start));

    fft<<<DATA_SIZE / 256, 256>>>(dev_middle, dev_data);

    cudaCheckReturn(hipEventRecord(stop));
    cudaCheckReturn(hipEventSynchronize(stop));
    cudaCheckKernel();

    cudaCheckReturn(hipEventElapsedTime(&time[0], start, stop));

    /*
      Scaling
    */
    cudaCheckReturn(hipMemcpy(middle, dev_middle, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyDeviceToHost));

    for (size_t i = 0; i < DATA_SIZE; i++) {
        float2 m = __half22float2(middle[i]);
        middle[i] = __floats2half2_rn(m.x / DATA_SIZE, m.y / DATA_SIZE);
    }    

    cudaCheckReturn(hipMemcpy(dev_middle, middle, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyHostToDevice));

    /*
      IFFT
    */
    cudaCheckReturn(hipDeviceSynchronize());
    cudaCheckReturn(hipEventRecord(start));

    ifft<<<DATA_SIZE / 256, 256>>>(dev_output, dev_middle);

    cudaCheckReturn(hipEventRecord(stop));
    cudaCheckReturn(hipEventSynchronize(stop));
    cudaCheckKernel();

    cudaCheckReturn(hipEventElapsedTime(&time[1], start, stop));

    /*
      Close
    */
    cudaCheckReturn(hipMemcpy(output, dev_output, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyDeviceToHost));

    cudaCheckReturn(hipHostFree(middle));

    cudaCheckReturn(hipFree(dev_output));
    cudaCheckReturn(hipFree(dev_middle));
    cudaCheckReturn(hipFree(dev_data));

    return time;
}
