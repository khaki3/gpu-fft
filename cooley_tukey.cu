#include "hip/hip_runtime.h"
#include "common.hu"
#include <hip/hip_cooperative_groups.h>

#define HIP_PI_F 3.141592654f

__device__ __forceinline__ DATA_TYPE mul(DATA_TYPE a, DATA_TYPE b)
{
    // (a_r b_r - a_i b_i), (a_r b_i + a_i b_r)
    DATA_TYPE m  = __floats2half2_rn(-1.0, 1.0);

    // (a_r b_r, a_i b_r)
    DATA_TYPE c = __hmul2(a, __low2half2(b));
    // (- a_i b_i, a_r b_i)
    DATA_TYPE s = __hmul2(m, __hmul2(__lowhigh2highlow(a), __high2half2(b)));

    return __hadd2(c, s);
}

__device__ __forceinline__ DATA_TYPE twiddle(DATA_TYPE a, size_t n, size_t block, size_t row)
{
    // todo: static
    float f = 2 * HIP_PI_F * block * row / n;
    return mul(a, __floats2half2_rn(cosf(f), - sinf(f)));
}

__global__ void fft(DATA_TYPE *data)
{
    cooperative_groups::grid_group g = cooperative_groups::this_grid();

    __shared__ DATA_TYPE sm[1024];
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;

    const float f_real[] = { 
        1.f,  1.f,  1.f,  1.f,
        1.f,  0.f, -1.f,  0.f,
        1.f, -1.f,  1.f, -1.f,
        1.f,  0.f, -1.f,  0.f
    };

    const float f_imag[] = {
        0.f,  0.f, 0.f,  0.f,
        0.f,  1.f, 0.f, -1.f,
        0.f,  0.f, 0.f,  0.f,
        0.f, -1.f, 0.f,  1.f
    };

    size_t idxl4 = threadIdx.x & 3 * 4;
    DATA_TYPE f_0 = __floats2half2_rn(f_real[idxl4 + 0], f_imag[idxl4 + 0]);
    DATA_TYPE f_1 = __floats2half2_rn(f_real[idxl4 + 1], f_imag[idxl4 + 1]);
    DATA_TYPE f_2 = __floats2half2_rn(f_real[idxl4 + 2], f_imag[idxl4 + 2]);
    DATA_TYPE f_3 = __floats2half2_rn(f_real[idxl4 + 3], f_imag[idxl4 + 3]);

    for (size_t n  = 4; n <= DATA_SIZE / 4; n *= 4) {
        if (n != 4) {
            g.sync();
        }

        size_t gblock = id / n;
        size_t offset = gblock * n;
        size_t block = gblock - (gblock / 4) * 4;
        size_t pos   = id - offset;
        size_t row   = pos / 4;
        size_t col   = pos - row * 4;

        // transpose, multiply by twiddle factor (W^(block, row))
        DATA_TYPE x = twiddle(data[offset + col * n + row], n, block, row);

        sm[threadIdx.x] = x;
        __syncwarp();

        // gemm
        data[id] = __hadd2(__hadd2(mul(sm[threadIdx.x - col + 0], f_0),
                                   mul(sm[threadIdx.x - col + 1], f_1)),
                           __hadd2(mul(sm[threadIdx.x - col + 2], f_2),
                                   mul(sm[threadIdx.x - col + 3], f_3)));
    }
}

std::vector<float> benchmark(DATA_TYPE *output,
                             DATA_TYPE *data,
                             hipEvent_t start, hipEvent_t stop)
{
    DATA_TYPE *dev_output, *dev_middle, *dev_data, *middle;
    std::vector<float> time(2);

    /*
      Setup
    */
    cudaCheckReturn(hipHostMalloc(&middle, DATA_SIZE * sizeof(DATA_TYPE)));

    cudaCheckReturn(hipMalloc(&dev_data,      DATA_SIZE * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipMalloc(&dev_middle,    DATA_SIZE * sizeof(DATA_TYPE)));
    cudaCheckReturn(hipMalloc(&dev_output,    DATA_SIZE * sizeof(DATA_TYPE)));

    cudaCheckReturn(hipMemcpy(dev_middle, data, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyHostToDevice));

    hipfftHandle plan;
    cufftCheckReturn(hipfftCreate(&plan));
    long long len = DATA_SIZE;
    size_t ws = 0;

    cufftCheckReturn(
        hipfftXtMakePlanMany(
            plan, 1,  &len,
            NULL, 1, 1, HIP_C_16F,
            NULL, 1, 1, HIP_C_16F,
            1, &ws, HIP_C_16F));

    /*
      FFT
    */
    cudaCheckReturn(hipDeviceSynchronize());
    cudaCheckReturn(hipEventRecord(start));

//    cufftCheckReturn(hipfftXtExec(plan, dev_data, dev_middle, HIPFFT_FORWARD));
    void *args[2];
    args[0] = dev_middle;
    args[1] = NULL;
    hipLaunchCooperativeKernel((void*)fft, dim3(DATA_SIZE / 1024, 1, 1), dim3(1024, 1, 1), args);

    cudaCheckReturn(hipEventRecord(stop));
    cudaCheckReturn(hipEventSynchronize(stop));
    cudaCheckKernel();

    cudaCheckReturn(hipEventElapsedTime(&time[0], start, stop));

    /*
      Scaling
    */
    cudaCheckReturn(hipMemcpy(middle, dev_middle, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyDeviceToHost));

    for (size_t i = 0; i < DATA_SIZE; i++) {
        float2 m = __half22float2(middle[i]);
        middle[i] = __floats2half2_rn(m.x / DATA_SIZE, m.y / DATA_SIZE);
    }    

    cudaCheckReturn(hipMemcpy(dev_middle, middle, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyHostToDevice));

    /*
      IFFT
    */
    cudaCheckReturn(hipDeviceSynchronize());
    cudaCheckReturn(hipEventRecord(start));

    cufftCheckReturn(hipfftXtExec(plan, dev_middle, dev_output, HIPFFT_BACKWARD));

    cudaCheckReturn(hipEventRecord(stop));
    cudaCheckReturn(hipEventSynchronize(stop));
    cudaCheckKernel();

    cudaCheckReturn(hipEventElapsedTime(&time[1], start, stop));

    /*
      Close
    */
    cufftCheckReturn(hipfftDestroy(plan));

    cudaCheckReturn(hipMemcpy(output, dev_output, DATA_SIZE * sizeof(DATA_TYPE),
                               hipMemcpyDeviceToHost));

    cudaCheckReturn(hipHostFree(middle));

    cudaCheckReturn(hipFree(dev_output));
    cudaCheckReturn(hipFree(dev_middle));
    cudaCheckReturn(hipFree(dev_data));

    return time;
}
